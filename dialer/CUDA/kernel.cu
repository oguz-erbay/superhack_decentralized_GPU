﻿
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 16  // Matris boyutu (NxN)

// CUDA kernel fonksiyonu
__global__ void matrixMultiply(int* A, int* B, int* C, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int sum = 0;
    if (row < n && col < n) {
        for (int i = 0; i < n; ++i) {
            sum += A[row * n + i] * B[i * n + col];
        }
        C[row * n + col] = sum;
    }
}

int main() {
    int size = N * N * sizeof(int);
    int A[N][N], B[N][N], C[N][N];  // Host matrisler

    // Matrisleri doldur (örnek veriler)
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            A[i][j] = i + j;
            B[i][j] = i - j;
            C[i][j] = 0;
        }
    }

    int *d_A, *d_B, *d_C;

    // Device (GPU) hafızası ayır
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Matrisleri GPU'ya kopyala
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // Kernel çağrısı (her blokta 16x16 thread)
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((N + threadsPerBlock.x - 1) / threadsPerBlock.x, 
                       (N + threadsPerBlock.y - 1) / threadsPerBlock.y);
    matrixMultiply<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

    // Sonucu GPU'dan geri al
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    // Sonucu ekrana yazdır (isteğe bağlı)
    printf("RESULT MATRIX:\n");
    for (int i = 0; i < N; ++i) {
        for (int j = 0; j < N; ++j) {
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    // GPU hafızasını temizle
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
